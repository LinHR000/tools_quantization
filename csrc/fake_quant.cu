#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDAGuard.h>

#include "cuda_compat.h"
#include "dispatch_utils.h"
#ifdef ENABLE_FP8_E5M2
#include "quant_utils.cuh"
#endif

#include <algorithm>
#include <cassert>
#include <map>
#include <vector>

#ifdef USE_ROCM
  #include <hip/hip_bf16.h>
  typedef __hip_bfloat16 __hip_bfloat16;
#endif

namespace vllm {

template<typename Tout>
__global__ void fake_quant_fp8_e5m2_kernel(
  const Tout* __restrict__ src_cache,
  Tout* __restrict__ dst_cache,
  const int64_t block_stride) {
  const int64_t block_idx = blockIdx.x;
  for (int i = threadIdx.x; i < block_stride; i += blockDim.x) {
    int64_t idx = block_idx * block_stride + i;
#ifdef ENABLE_FP8_E5M2
    uint8_t tmp_val = fp8_e5m2_unscaled::vec_conversion<uint8_t,Tout>(src_cache[idx]);
    dst_cache[idx] = fp8_e5m2_unscaled::vec_conversion<Tout, uint8_t>(tmp_val);
#else
    assert(false);
#endif
  }
}

} // namespace vllm


#define CALL_FAKE_QUANT_FP8_E5M2(Tout)                                   \
  vllm::fake_quant_fp8_e5m2_kernel<Tout><<<grid, block, 0, stream>>>(    \
    reinterpret_cast<Tout*>(src_cache.data_ptr()),                       \
    reinterpret_cast<Tout*>(dst_cache.data_ptr()),                       \
    block_stride);

torch::Tensor fake_quant_fp8_e5m2(
  torch::Tensor& src_cache,
  torch::Tensor& dst_cache)
{
  int64_t num_blocks = src_cache.size(0);
  int64_t block_stride = src_cache.stride(0);

  dim3 grid(num_blocks);
  dim3 block(std::min(block_stride, int64_t(512)));
  const hipStream_t stream = at::cuda::getCurrentCUDAStream();

  if (src_cache.dtype() == at::ScalarType::Half) {
    CALL_FAKE_QUANT_FP8_E5M2(uint16_t);
  }else{
    TORCH_CHECK(false, "Unsupported data type:  ", src_cache.dtype());
  }
  return dst_cache;
}


